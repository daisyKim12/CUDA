#include "hip/hip_runtime.h"
/* 
    GPU는 Pinned mem만 접근할 수 있다.
    dGPU의 경우 Pinned mem에서 데이터를 복사하고 iGPU는 Pinned mem에서 데이터를 device 메모리 공간으로 옮긴다.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "9_util/common.h"
#include </usr/local/cuda/include/hip/hip_runtime.h>
#include </usr/local/cuda/include/hip/hip_runtime_api.h>

void initialData(float* in, const long long int size)
{
    for (int i = 0; i < size; i++)
        in[i] = (rand() & 0xFF) / 10.f;
}

void sumMatrixOnHost(float *A, float *B, float *C, const long long int nx, const long long int ny)
{
    float* ia = A;
    float* ib = B;
    float* ic = C;

    for (int iy = 0; iy < ny; iy++) {
        for (int ix = 0; ix < nx; ix++) {
            ic[ix] = ia[ix] + ib[ix];
        }

        ia += nx;
        ib += nx;
        ic += nx;
    }
}

void getFullBandwidthOnHost(float *A, const long long int nx, const long long int ny)
{
    
    float* ia = A;
    float temp = 0;

    for(long long int i = 0; i < nx * ny; i++)
    {
        temp = ia[i];
        temp = temp + 1;
        ia[i] = temp;
    }
}

void checkResult(float* hostRef, float* gpuRef, const int size)
{
    double epsilon = 1.0e-8;

    for (int i = 0; i < size; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
            printf("different on %dth element: host %f gpu %f\n", i, hostRef[i], gpuRef[i]);
            break;
        }
    }
  
}


__global__
void sumMatrixOnGPU(float* A, float* B, float* C, const long long int width, const long long int height)
{
    unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int idx = row * width + col;

    if (col < width && row < height)
        C[idx] = A[idx] + B[idx];
}

int main(int argc, char** argv)
{
    // setup device
    int nDevice = 0;

    hipGetDeviceCount(&nDevice);
    printf("Number of device: %d\n", nDevice);

    for(int i = 0; i<nDevice; i++) {
        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, i));
        
        printf("Device Number: %d\n", 0);
        printf("  Device name: %s\n", prop.name);
        printf("  Device L2 cache size: %d\n", prop.l2CacheSize);
        printf("  Device Shared mem per block: %ld\n", prop.sharedMemPerBlock);

        printf("  Memory Clock Rate (KHz): %d\n",
            prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
            prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
            2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }

    // setup size of matrix
    long long int nx, ny;
    long long int power = 12;
    if (argc > 1)
        power = atoi(argv[1]);
    nx = ny = 1 << power;

    long long int nxy = nx * ny;
    size_t nBytes = nxy * sizeof(float);

// using hipMemcpy
    printf("part 1: using hipMemcpy\n");
    printf("Matrix size: nx %llu ny %llu\n", nx, ny);

    float *M_d, *N_d,  *S_d;
    float *M_h = new float[nBytes];
    float *N_h = new float[nBytes];
    float *S_h = new float[nBytes];
    CUDA_CHECK(hipMalloc((void**)&M_d, nBytes));
    CUDA_CHECK(hipMalloc((void**)&N_d, nBytes));
    CUDA_CHECK(hipMalloc((void**)&S_d, nBytes));
    
    double start, finish;
    GET_TIME(start);
    initialData(M_h, nxy);
    initialData(N_h, nxy);
    GET_TIME(finish);
    printf("initialization: \t %f sec\n", finish - start);

    GET_TIME(start);
    hipMemcpy(M_d, M_h, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(N_d, N_h, nBytes, hipMemcpyHostToDevice);
    CUDA_CHECK(hipDeviceSynchronize());
    GET_TIME(finish);
    printf("memcpy host -> dev: \t %f sec\n", finish - start);

    GET_TIME(start);
    sumMatrixOnHost(M_h, N_h, S_h, nx, ny);
    GET_TIME(finish);
    printf("sumMatrix on host:\t %f sec\n", finish - start);
    printf("Utilized CPU bandwidth (GB/s): %f\n", (3 * nBytes) / 1.0e9 / (finish - start));

    GET_TIME(start);
    getFullBandwidthOnHost(M_h, nx, ny);
    GET_TIME(finish);
    printf("getFullBandwidth on host:\t %f sec\n", finish - start);
    printf("Utilized CPU bandwidth (GB/s): %f\n", (2 * nBytes) / 1.0e9 / (finish - start));

    // invode kernel at host side
    int dimX = 32;
    int dimY = 32;
    dim3 blocks(dimX, dimY);
    dim3 grids((nx + blocks.x - 1) / blocks.x, (ny + blocks.y - 1) / blocks.y);
    
    GET_TIME(start);
    sumMatrixOnGPU<<<grids, blocks>>>(M_d, N_d, S_d, nx, ny);
    hipDeviceSynchronize();
    GET_TIME(finish);

    printf("sumMatrix on gpu :\t %f sec <<<(%d,%d), (%d,%d)>>>\n", (finish - start), grids.x, grids.y, blocks.x, blocks.y);
    printf("Utilized GPU bandwidth (GB/s): %f\n", (3 * nBytes) / 1.0e9 / (finish - start));

    GET_TIME(start);
    CUDA_CHECK(hipMemcpy(S_h, S_d, nBytes, hipMemcpyDeviceToHost))
    CUDA_CHECK(hipDeviceSynchronize());
    GET_TIME(finish);
    printf("memcpy host -> dev: \t %f sec\n", finish - start);

    CUDA_CHECK(hipFree(M_d));
    CUDA_CHECK(hipFree(N_d));
    CUDA_CHECK(hipFree(S_d));
    delete[] M_h;
    delete[] N_h;
    delete[] S_h;

    return 0;
}


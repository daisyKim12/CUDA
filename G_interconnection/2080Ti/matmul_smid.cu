
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <chrono>
#include <math.h>
#include </usr/local/cuda/include/cuda.h>
#include </usr/local/cuda/include/cuda_runtime_api.h>
#include <iostream>

#define NUM 10
#define SKEW 1

#define TILE_WIDTH 16
#define WIDTH 8192     //65536 main memory shortage

__device__ 
uint get_smid(void) {

     uint ret;

     asm("mov.u32 %0, %smid;" : "=r"(ret) );

     return ret;

}

__device__
uint get_nsmid(void){

    uint ret;

    asm("mov.u32 %0, %%nsmid;" : "=r"(ret));

    return ret;

}

__global__ void matmul(float* M, float* N, float* R, const long long int width)
{   
    // check sm id
    if(threadIdx.x == 0){
        
        //printf("number of sm allocated for kernel: %d \n", get_nsmid());
        printf("%d ", get_smid());
    }

    __shared__ float sub_tile_M[TILE_WIDTH * 4][TILE_WIDTH];
    __shared__ float sub_tile_N[TILE_WIDTH * 1][TILE_WIDTH];

    long long int tx = threadIdx.x;  long long int ty = threadIdx.y;
    long long int bx = blockIdx.x;   long long int by = blockIdx.y;
    
    long long int col = tx + TILE_WIDTH * bx;
    long long int row = ty + (TILE_WIDTH * 4) * by;

    float acc_1 = 0;
    float acc_2 = 0;
    float acc_3 = 0;
    float acc_4 = 0;


    for(int idx = 0 ; idx < width/TILE_WIDTH; idx ++) {
        //load top sqaure of M
        sub_tile_M[ty][tx] = M[row * width + idx * TILE_WIDTH + tx];
        sub_tile_M[ty + TILE_WIDTH][tx] = M[(row + TILE_WIDTH)* width + idx * TILE_WIDTH + tx];
        sub_tile_M[ty + TILE_WIDTH*2][tx] = M[(row + TILE_WIDTH*2)* width + idx * TILE_WIDTH + tx];
        sub_tile_M[ty + TILE_WIDTH*3][tx] = M[(row + TILE_WIDTH*3)* width + idx * TILE_WIDTH + tx];


        // load single square of N
        sub_tile_N[ty][tx] = N[(idx * TILE_WIDTH + ty) * width + col];

        __syncthreads();

        // compute
        for(int k = 0; k < TILE_WIDTH; k++) {
            acc_1 += sub_tile_M[ty][k] * sub_tile_N[k][tx];
            acc_2 += sub_tile_M[ty + TILE_WIDTH][k] * sub_tile_N[k][tx];
            acc_3 += sub_tile_M[ty + TILE_WIDTH*2][k] * sub_tile_N[k][tx];
            acc_4 += sub_tile_M[ty + TILE_WIDTH*3][k] * sub_tile_N[k][tx];

        }
        

        __syncthreads();

    }

    R[row * width + col] = acc_1;
    R[(row + TILE_WIDTH) * width + col] = acc_2;
    R[(row + TILE_WIDTH * 2) * width + col] = acc_3;
    R[(row + TILE_WIDTH * 3) * width + col] = acc_4;


}

double run_matmul(float *M, float *N,  float *out, long long int width, int tile_width) {

    long long int height = width;

    dim3 dimGrid4(width/TILE_WIDTH, (height/TILE_WIDTH)/4, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
    matmul<<<dimGrid4, dimBlock>>>(M, N, out, width);
    hipDeviceSynchronize();
    std::chrono::duration<double>sec = std::chrono::system_clock::now() - start;

    return sec.count();
}

// initializing array size N with random float between 0 and 100
void init_array(float *arr, long long int N, int seed) {
    
    srand(seed);

    for(long long int i = 0; i<N; i++){
        arr[i] = ((float)rand() / RAND_MAX) * 100.0;
    }
}

// display partial array size m
void print_array(float *arr, int m) {
    for(int i = 0; i < m; i++) {
        std::cout << arr[i]<< " ";
    }
    std::cout << "...\n" << std::endl;
}

int main(int argc, char *argv[]) {

    
    long long int width = WIDTH;
    // if(argc > 1) {
    //     width = (1 << atoi(argv[1]));
    // }
    long long int tile_width = TILE_WIDTH;
    long long int total_size = width * width;

    float *M_h = new float[total_size];
    float *N_h = new float[total_size];
    float *result_h = new float[total_size];
    float *M_d, *N_d, *result_d;

    double run_time = 0;

    init_array(M_h, total_size, 8811);
    init_array(N_h, total_size, 9700);

    // memory allocation
    hipMalloc((void**)&M_d, total_size * sizeof(float));
    hipMalloc((void**)&N_d, total_size * sizeof(float));
    hipMalloc((void**)&result_d, total_size * sizeof(float));
    // copy host memory to device
    hipMemcpy(M_d, M_h, total_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(N_d, N_h, total_size * sizeof(float), hipMemcpyHostToDevice);
    
    std::cout << " running ... >\n";
    hipError_t cudaErr = hipGetLastError();
    std::cerr << "CUDA error: " << hipGetErrorString(cudaErr) << "\n" << std::endl;    
    run_time = run_matmul(M_d, N_d, result_d, width, tile_width);
    cudaErr = hipGetLastError();
    std::cerr << "CUDA error: " << hipGetErrorString(cudaErr) << "\n" << std::endl;    

    // copy result to host memory
    hipMemcpy(result_h, result_d, total_size * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "\nhost memory: result_h" << std::endl;
    print_array(result_h, NUM);

    std::cout << "\n--------------------result------------------\n" << std::endl;
    std::cout << "time: " << run_time <<"seconds"<< std::endl;
    
    hipFree(M_d); hipFree(N_d); hipFree(result_d);
    delete[] M_h; delete[] N_h; delete[] result_h;

    return 0;
}
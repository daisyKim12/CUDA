#include "hip/hip_runtime.h"
/* Reverse Engineering GPC Organization */

#include "9_util/common.h"

#define TPC_PER_GPC 6
#define MAX_SM 68               // SM   index 0~67
#define MAX_TPC 34              // TPC  index 0_33
#define PRINT_NUM 10

void initialData(int* in, const int size)
{
    for (int i = 0; i < size; i++)
        in[i] = rand() & 0xFF;
}

void checkArr(int* in, const int num)
{
    for(int i = 0; i< num; i++) {
        std::cout << in[i] << " ";
    }
    std::cout << "\n";
}


__device__ 
uint get_smid(void) {

     uint ret;

     asm("mov.u32 %0, %smid;" : "=r"(ret) );

     return ret;

}

__device__
bool is_in(uint a, uint * arr) {
    for(int i = 0; i < TPC_PER_GPC; i++)
        if(a == arr[i] * 2)
            return true;
    return false;
}

/* 
The goal here is to evaluate a memory-intensive program that continuously accesses the L2 cache 
(and bypass L1 cache such that the interconnect is accessed). 
The code does sequential memory write access and ensures that all memory partitions 
-(and corresponding L2 cache) are accessed by the SM.

To identify which SMs are co-located within each GPC, we activate one SM in each of the 6 TPCs
- i.e., 6 SMs in total. 
Using a similar approach as before, we always activate TPC0 and then vary the TPC that is selected to run concurrently with TPC0. 
We use only one SM from each TPC. 5 other TPCs are randomly selected or made active and we run the evaluation 200 times. 
Unlike the TPC channel evaluation where we only selected 2 SMs, 6 SMs are needed for this evaluation because of the bandwidth
*/


__global__ 
void memory_write_test(int* A_h, int* B_h, int array_size, uint fixed_sm_id, uint *config_sm_id)
{
    /* 
    sm_id: current sm
    config_sm_id: the sm number that i want to activate to see the influence to sm0 execution time
    */
    
    uint sm_id = get_smid();

    int thread_idx = threadIdx.x;
    int block_size = blockDim.x;

    int amount = array_size / block_size;       // size of data for a single thread
    int base = amount * thread_idx;             // base index for a single thread

    // if current sm is sm0
    if (sm_id == fixed_sm_id * 2) {
        // all thread in sm0 will write to A_h-> sequential write 
        for(int i = 0; i<amount; i++) {
            A_h[base + i] = thread_idx;
        }
    }
    // if current sm is the sm that i want to check
    else if(is_in(sm_id, config_sm_id)) {
        // all thread in config sm will write to B_h-> sequential write
        for(int i = 0; i<amount; i++) {
            B_h[base + i] = thread_idx;
        }
    }

}


int main(int argc, char** argv) {

    if( argc <= 1)
        return 0;

    int fixed_sm_id = atoi(argv[1]);

    double start, finish;
    
    int long long nx;
    int power = 14;
    nx = 1 << power;
    size_t nBytes = nx * sizeof(int);

    int *A_h = new int[nx];
    int *B_h = new int[nx];
    int *A_d, *B_d;
    CUDA_CHECK(hipMalloc((void**)&A_d, nBytes));
    CUDA_CHECK(hipMalloc((void**)&B_d, nBytes));
    
    // initialData(A_h, nx);
    // initialData(B_h, nx);

    // checkArr(A_h, PRINT_NUM);
    // checkArr(B_h, PRINT_NUM);

    hipMemcpy(A_d, A_h, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, nBytes, hipMemcpyHostToDevice);

    double *time = new double[MAX_SM];

    double max = -1;
    double avg = 0;
    int max_idx = -1;

    // warmup kernel
    // memory_write_test<<<nx/8, 8>>>(A_d, B_d, nx, fixed_sm_id, 0);
    // CUDA_CHECK(hipDeviceSynchronize());
    // memory_write_test<<<nx/8, 8>>>(A_d, B_d, nx, fixed_sm_id, 0);
    // CUDA_CHECK(hipDeviceSynchronize());

    uint check_sm_list[5] = {3,6,8,12,33};
    GET_TIME(start);
    memory_write_test<<<nx/8, 8>>>(A_d, B_d, nx, fixed_sm_id, check_sm_list);
    CUDA_CHECK(hipDeviceSynchronize());
    GET_TIME(finish);
    double duration = finish - start;
    std::cout << duration << "\n";

    // time[i] = duration;
    // if(max < duration) {
    //     max = duration;
    //     max_idx = i;
    // }
    // avg = avg + duration;
    

    // for(int i = 0; i<MAX_SM; i++) {
    //     std::cout << i << ": " << time[i] << "\n";
    // }
    //     std::cout << "\n";

    avg = avg / MAX_SM;
    std::cout << "Maximum: (" << max_idx << ")" << max << "\n";
    std::cout << "Average: " << avg << "\n";
    
    CUDA_CHECK(hipMemcpy(A_h, A_d, nBytes, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(B_h, B_d, nBytes, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());                                //must add cuda device sync to get updated array
    
    // checkArr(A_h, PRINT_NUM);
    // checkArr(B_h, PRINT_NUM);

    delete[] A_h;
    delete[] B_h;
    CUDA_CHECK(hipFree(A_d));
    CUDA_CHECK(hipFree(B_d));

    return 0;
}
#include "hip/hip_runtime.h"
/* Reverse Engineering TPC Organization */

#include "9_util/common.h"

#define MAX_SM 68
#define PRINT_NUM 16384

void initialData(int* in, const int size)
{
    for (int i = 0; i < size; i++)
        in[i] = rand() & 0xFF;
}

void checkArr(int* in, const int num)
{
    for(int i = 0; i< num; i++) {
        std::cout << in[i] << " ";
    }
    std::cout << "\n";
}


__device__ 
uint get_smid(void) {

     uint ret;

     asm("mov.u32 %0, %smid;" : "=r"(ret) );

     return ret;

}

/* 
The goal here is to evaluate a memory-intensive program that continuously accesses the L2 cache 
(and bypass L1 cache such that the interconnect is accessed). 
The code does sequential memory write access and ensures that all memory partitions 
-(and corresponding L2 cache) are accessed by the SM.

We execute this synthetic code concurrently on SM0 and one other SM in the GPU, 
i.e., only two SMs are active.
*/


__global__ 
void memory_write_test(int* A_h, int* B_h, int array_size, uint fixed_sm_id, uint config_sm_id)
{
    /* 
    sm_id: current sm
    config_sm_id: the sm number that i want to activate to see the influence to sm0 execution time
    */

    uint sm_id = get_smid();
    int thread_idx = threadIdx.x;
    int block_size = blockDim.x;

    int amount = array_size / block_size;       // size of data for a single thread
    int base = amount * thread_idx;             // base index for a single thread

    // if current sm is sm0
    if (sm_id == fixed_sm_id) {
        // all thread in sm0 will write to A_h-> sequential write 
        for(int i = 0; i<amount; i++) {
            A_h[base + i] = thread_idx;
            //A_h[base + i] = sm_id;
        }
    }
    // if current sm is the sm that i want to check
    else if(sm_id == config_sm_id) {
        // all thread in config sm will write to B_h-> sequential write
        for(int i = 0; i<amount; i++) {
            B_h[base + i] = thread_idx;
            //B_h[base + i] = sm_id;
        }
    }

}


int main(int argc, char** argv) {

    if( argc <= 1)
        return 0;

    int fixed_sm_id = atoi(argv[1]);

    double start, finish;
    
    int long long nx;
    int power = 14;
    nx = 1 << power;
    size_t nBytes = nx * sizeof(int);

    int *A_h = new int[nx];
    int *B_h = new int[nx];
    int *A_d, *B_d;
    CUDA_CHECK(hipMalloc((void**)&A_d, nBytes));
    CUDA_CHECK(hipMalloc((void**)&B_d, nBytes));
    
    // initialData(A_h, nx);
    // initialData(B_h, nx);

    // checkArr(A_h, PRINT_NUM);
    // checkArr(B_h, PRINT_NUM);

    hipMemcpy(A_d, A_h, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, nBytes, hipMemcpyHostToDevice);

    double *time = new double[MAX_SM];

    double max = -1;
    double avg = 0;
    int max_idx = -1;

    // warmup kernel
    memory_write_test<<<nx/8, 8>>>(A_d, B_d, nx, fixed_sm_id, 0);
    CUDA_CHECK(hipDeviceSynchronize());
    memory_write_test<<<nx/8, 8>>>(A_d, B_d, nx, fixed_sm_id, 0);
    CUDA_CHECK(hipDeviceSynchronize());

    for(int i = 0; i<MAX_SM; i++) {
        GET_TIME(start);
        memory_write_test<<<nx/8, 8>>>(A_d, B_d, nx, fixed_sm_id, i);
        CUDA_CHECK(hipDeviceSynchronize());
        GET_TIME(finish);
        double duration = finish - start;
        time[i] = duration;
        if(max < duration) {
            max = duration;
            max_idx = i;
        }
        avg = avg + duration;
    }

    for(int i = 0; i<MAX_SM; i++) {
        std::cout << i << ", " << time[i] << "\n";
    }
        std::cout << "\n";

    avg = avg / MAX_SM;
    std::cout << "Maximum: (" << max_idx << ")" << max << "\n";
    std::cout << "Average: " << avg << "\n";
    
    CUDA_CHECK(hipMemcpy(A_h, A_d, nBytes, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(B_h, B_d, nBytes, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());                                //must add cuda device sync to get updated array
    
    // checkArr(A_h, PRINT_NUM);
    // checkArr(B_h, PRINT_NUM);

    delete[] A_h;
    delete[] B_h;
    CUDA_CHECK(hipFree(A_d));
    CUDA_CHECK(hipFree(B_d));

    return 0;
}